
#include <hip/hip_runtime.h>
#include <iostream>
#include <string> // stof, stoi
#include <chrono>
#include <ctime>
#include <fstream>
#include <cmath>
#include <iomanip>

#include <stdio.h>
#include <math.h>
#include <assert.h>

// CUDA libraries
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

// function to integrate: sin(x)/x

struct mem_t {
    double* ptr;
    size_t numItems;
    size_t numBytes;

    mem_t(): ptr(NULL), numItems(0), numBytes(0) {};
};

// monte carlo ; each CUDA thread
__global__ void integrate(double a, double b, int numSamples, double* work, double* results, int bx, int by) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    int id = iy*bx + ix;
    
    //std::default_random_engine randEngine;
    //randEngine.seed(id);
    //std::uniform_real_distribution<double> randGen(a, b);
    hiprandState_t state;
    hiprand_init(id, 0, 0, &state); // curand_init(seed, sequence number, offset, &state)

    for(int i=0; i<numSamples; i++) {
        //double x = randGen(randEngine);
		// (rand() % (upper - lower + 1)) + lower; 
        double x = hiprand_uniform_double(&state);
        x *= (b - a + 0.999999);
        x += a;
        double result = sin(x)/x;
        work[id] += (result * !isnan(result)); // if nan, it will simply add zero ; reduces branches
    }
    
    //synchronize threads within block
    __syncthreads();

    // sum up the values in each row
    if(threadIdx.x == 0) { // if thread column is left-most
        for(int x=1; x<bx; x++) { // sum up all the partial sums in this row
            int index = iy*bx + (x + blockIdx.x * blockDim.x);
            work[id] += work[index];
        }
    }   
 
    //synchronize threads within block
    __syncthreads();

    // sum up each row's sum into the results
    if(threadIdx.x == 0 && threadIdx.y == 0) {
        for(int y=0; y<by; y++) {
            int index = (y + blockIdx.y * blockDim.y)*bx + ix;
            results[id % (bx*by)] += work[index];
        }
    }

}

int main(int argc, char* argv[]) {
    
    if(argc < 5) {
        printf("./integrate lowerLimit, upperLimit, numSamples, numThreads\n");
        exit(1);
    }
   
    double a = std::stof(argv[1]); // lower limit
    double b = std::stof(argv[2]); // upper limit
    int numSamples = std::stoi(argv[3]); 
    int numThreads = std::stoi(argv[4]);

    std::chrono::time_point<std::chrono::system_clock> start, end;
    start = std::chrono::system_clock::now(); 
   
    // define grid and block structure
    const int bx = 32;
    const int by = 32;
    const int numBlocks = ceil(numThreads / (bx*by));
    dim3 block(bx, by); // 2D block of 32x32 = 1,024 threads per block
    dim3 grid(numBlocks); // 1D grid
    printf("a=%.2f,b=%.2f,numSamples=%i,numThreads=%i,numBlocks=%i\n", a, b, numSamples, numThreads, numBlocks);
    printf("block.x=%u, block.y=%u\n", block.x, block.y);
 
    // set up memory for results
    //double* d_results;
    //size_t numBytes = numBlocks * sizeof(double);
    mem_t d_results;
    d_results.numItems = numBlocks;
    d_results.numBytes = d_results.numItems * sizeof(double);
    hipMalloc((void**)&d_results.ptr, d_results.numBytes); // each block will compute sum ; host will add up the block sums
    //cudaMalloc((void**)&d_results, numBytes); // each block will compute sum ; host will add up the block sums

    // set up memory for work buffer ; this buffer does not have to be sent back to the host
    mem_t d_work;
    d_work.numItems = (bx*by) * numBlocks;
    d_work.numBytes = d_work.numItems * sizeof(double);
    hipMalloc((void**)&d_work.ptr, d_work.numBytes); // each block will compute sum ; host will add up the block sums

    // launch CUDA kernel 
    integrate<<<grid, block>>>(a, b, numSamples/numThreads, d_results.ptr, d_work.ptr, bx, by);

    // copy results from device to host
    double* h_results = (double*) malloc(d_results.numBytes);
    hipMemcpy(h_results, d_results.ptr, d_results.numBytes, hipMemcpyDeviceToHost); 
   
    // compute the sum of each block
    double integral = 0.0;
    for(int i=0; i<numBlocks; i++) {
        integral += h_results[i];
    }

    // free device memory
    hipFree(d_results.ptr);
    hipFree(d_work.ptr);
    
    // free host memory
    free(h_results);

    end = std::chrono::system_clock::now();
    std::chrono::duration<double> elapsed_sec = end - start; 
    std::cout << numThreads << " threads: " << " numBlocks " << numBlocks << " Result: " << std::setprecision(10) << integral << "; Elapsed time: " << elapsed_sec.count() << "s" << std::endl;
    //std::cout << std::setprecision(10) << integral << std::endl;

    // write to csv file
    std::string csvfile_name = std::to_string((int)abs(a)) + "-" + std::to_string((int)abs(b)) + "-" + std::to_string(numSamples) + "-" + std::to_string(numThreads) + ".csv";
    std::ofstream csvfile;
    csvfile.open(csvfile_name);
    // write csv header
    csvfile << "a,b,numSamples,integral,elapsed(sec),method,numThreads\n";
    
    csvfile << std::to_string((int)a) << ","
            << std::to_string((int)b) << ","
            << std::to_string(numSamples) << ","
            << std::to_string(integral) << ","
            << elapsed_sec.count() << ","
            << "monte-carlo" << ','
            << std::to_string(numThreads) << "\n";
    
    csvfile.close();
}
