
#include <hip/hip_runtime.h>
#include <iostream>
#include <string> // stof, stoi
#include <chrono>
#include <ctime>
#include <fstream>
#include <cmath>
#include <iomanip>

#include <stdio.h>
#include <math.h>
#include <assert.h>

// CUDA libraries
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define DEBUG 0

// function to integrate: sin(x)/x

struct mem_t {
    double* ptr;
    size_t numItems;
    size_t numBytes;

    mem_t(): ptr(NULL), numItems(0), numBytes(0) {};
};

// monte carlo ; each CUDA thread
__global__ void integrate(double a, double b, int numSamples, double* work, double* results, int bx, int by) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    int id = iy*bx + ix;
    
    hiprandState_t state;
    hiprand_init(id, 0, 0, &state); // curand_init(seed, sequence number, offset, &state)

    for(int i=0; i<numSamples; i++) {
		// (rand() % (upper - lower + 1)) + lower; 
        double x = hiprand_uniform_double(&state);
        x *= (b - a + 0.999999);
        x += a;
        double result = sin(x)/x;
        result = result * (isnan(result) == 0);
        work[id] += result; // if nan, it will simply add zero ; reduces branches
    }
    
    //synchronize threads within block
    __syncthreads();

    #if DEBUG
    if(id==0) {
        for(int i=0; i<bx*by; i++) {
            if(i%bx==0 && i!=0) printf("\n");
            printf("%.2f ", work[i]);
        }
        printf("\n");
    }

    __syncthreads();
    #endif
    
    // sum up the values in each row
    if(threadIdx.x == 0) { // if thread column is left-most
        for(int x=1; x<bx; x++) { // sum up all the partial sums in this row
            int index = iy*bx + (x + blockIdx.x * blockDim.x);
            work[id] += work[index];
        }
    }   
 
    //synchronize threads within block
    __syncthreads();
   
    #if DEBUG 
    if(id==0) {
        printf("After row sums computed\n");
        for(int i=0; i<bx*by; i++) {
            if(i%bx==0 && i!=0) printf("\n");
            printf("%.2f ", work[i]);
        }
        printf("\n");
    }

    __syncthreads();
    #endif

    // sum up each row's sum into the results
    if(threadIdx.x == 0 && threadIdx.y == 0) {
        for(int y=0; y<by; y++) {
            int index = (y + blockIdx.y * blockDim.y)*bx + ix;
            results[blockIdx.x] += work[index];
            #if DEBUG
            printf("id=%i, writing to results[%u]=%.2f, added=%.2f\n", id, blockIdx.x, results[blockIdx.x], work[index]);
            #endif
        }
    }
     
}

int main(int argc, char* argv[]) {
    
    if(argc < 5) {
        printf("./integrate lowerLimit, upperLimit, numSamples, numThreads\n");
        exit(1);
    }
   
    double a = std::stof(argv[1]); // lower limit
    double b = std::stof(argv[2]); // upper limit
    int numSamples = std::stoi(argv[3]); 
    int numThreads = std::stoi(argv[4]);

    std::chrono::time_point<std::chrono::system_clock> start, end;
    start = std::chrono::system_clock::now(); 
   
    // define grid and block structure ; largest possible block can hold 1,024 threads (32x32)
    int bx = 32;
    int by = 32;
    {
        int minLength = sqrt(numThreads);
        if(minLength <= 32) {
            bx = minLength;
            by = minLength;
        }
    }
    const int numBlocks = ceil(numThreads / (bx*by));
    dim3 block(bx, by);
    dim3 grid(numBlocks); // 1D grid
 
    // set up memory for results
    mem_t d_results;
    d_results.numItems = numBlocks;
    d_results.numBytes = d_results.numItems * sizeof(double);
    hipMalloc((void**)&d_results.ptr, d_results.numBytes); // each block will compute sum ; host will add up the block sums

    // set up memory for work buffer ; this buffer does not have to be sent back to the host
    mem_t d_work;
    d_work.numItems = (bx*by) * numBlocks;
    d_work.numBytes = d_work.numItems * sizeof(double);
    hipMalloc((void**)&d_work.ptr, d_work.numBytes); // each block will compute sum ; host will add up the block sums

    // launch CUDA kernel 
    int samplesPerThread = numSamples / ((bx*by) * numBlocks);
    integrate<<<grid, block>>>(a, b, samplesPerThread, d_work.ptr, d_results.ptr, bx, by);

    #if DEBUG
    printf("a=%.2f,b=%.2f,numSamples=%i,numThreads=%i,samplesPerThread=%i,numBlocks=%i\n", a, b, numSamples, numThreads, samplesPerThread, numBlocks);
    printf("block.x=%u, block.y=%u\n", block.x, block.y);
    #endif
    
    // copy results from device to host
    double* h_results = (double*) malloc(d_results.numBytes);
    hipMemcpy(h_results, d_results.ptr, d_results.numBytes, hipMemcpyDeviceToHost); 
   
    // compute the sum of each block
    double integral = 0.0;
    for(int i=0; i<numBlocks; i++) {
        #if DEBUG
        printf("Block %i: sum=%.2f\n", i, h_results[i]);
        #endif
        integral += h_results[i];
    }
    integral = abs(b-a) * (integral/(samplesPerThread * (bx*by) * numBlocks)); // sum/numSamples
    
    // free device memory
    hipFree(d_results.ptr);
    hipFree(d_work.ptr);
    
    // free host memory
    free(h_results);

    end = std::chrono::system_clock::now();
    std::chrono::duration<double> elapsed_sec = end - start; 
    std::cout << std::setprecision(10) << integral << std::endl;

    #if DEBUG
    std::cout << numThreads << " threads: " << " numBlocks " << numBlocks << ", Result: " << std::setprecision(10) << integral << "; Elapsed time: " << elapsed_sec.count() << "s" << std::endl;
    #endif    

    // write to csv file
    std::string csvfile_name = std::to_string((int)abs(a)) + "-" + std::to_string((int)abs(b)) + "-" + std::to_string(numSamples) + "-" + std::to_string(numThreads) + ".csv";
    std::ofstream csvfile;
    csvfile.open(csvfile_name);
    // write csv header
    csvfile << "a,b,numSamples,integral,elapsed(sec),method,block,numBlocks,numThreads\n";
    
    csvfile << std::to_string((int)a) << ","
            << std::to_string((int)b) << ","
            << std::to_string(numSamples) << ","
            << std::to_string(integral) << ","
            << elapsed_sec.count() << ","
            << "monte-carlo" << ','
            << "(" << bx << ";" << by << ")" << ","
            << numBlocks << ","
            << std::to_string(numThreads) << "\n";
    
    csvfile.close();
}
