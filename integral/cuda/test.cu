
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <ctime>

#include <stdio.h>
#include <math.h>
#include <assert.h>

__global__ void helloFromGPU() {
    printf("Hello from GPU!\n");
}

int main() {
    
    std::chrono::time_point<std::chrono::system_clock> start, end;
    start = std::chrono::system_clock::now(); 
   
    // cuda function here
    helloFromGPU<<<1,10>>>();
 
    end = std::chrono::system_clock::now();
    std::chrono::duration<double> elapsed_sec = end - start;
    printf("%.2f seconds\n", elapsed_sec);

    // cleans up device resources
    hipDeviceReset(); 
}
